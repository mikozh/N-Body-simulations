#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define HOSTLEN 50


// Initial conditions
void initCoord(float *rA, float *vA, float *fA, \
               float initDist, int nBod, int nI);

// Forces acting on each body
__global__ void forces(float *rA, float *fA, int nBod);

// Calculate velocities and update coordinates
__global__ void integration(float *rA, float *vA, float *fA, int nBod);

int main(int argc, const char * argv[]) {
   int const nI = 32;               // Number of bodies in X, Y and Z directions
   int const nBod = nI*nI*nI;       // Total Number of bodies
   int const maxIter = 20;          // Total number of iterations (time steps)
   float const initDist = 1.0;      // Initial distance between the bodies
   float *rA, *rA_d;                // Coordinates
   float *vA, *vA_d;                // Velocities
   float *fA, *fA_d;                // Forces
   float time;
   int iter;
   hipDeviceProp_t devProp;
   hipEvent_t start, stop;

   rA = (float*)malloc(3*nBod*sizeof(float));
   fA = (float*)malloc(3*nBod*sizeof(float));
   vA = (float*)malloc(3*nBod*sizeof(float));

   hipMalloc((void**)&rA_d, 3*nBod*sizeof(float));
   hipMalloc((void**)&vA_d, 3*nBod*sizeof(float));
   hipMalloc((void**)&fA_d, 3*nBod*sizeof(float));

   hipEventCreate(&start);
   hipEventCreate(&stop);

   hipGetDeviceProperties(&devProp, 0);
   printf("Name of CUDA GPU: %s\n",devProp.name);

   // Setup initial conditions
   initCoord(rA, vA, fA, initDist, nBod, nI);

   hipEventRecord(start, 0);

   hipMemcpy(rA_d, rA, 3*nBod*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(vA_d, vA, 3*nBod*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(fA_d, fA, 3*nBod*sizeof(float), hipMemcpyHostToDevice);

   // Main loop
   for ( iter = 0; iter < maxIter; iter++ ) {
      forces<<<nBod/512, 512>>>(rA_d, fA_d, nBod);

      integration<<<3*nBod/512, 512>>>(rA_d, vA_d, fA_d, nBod);
   }

   hipMemcpy(rA, rA_d, 3*nBod*sizeof(float), hipMemcpyDeviceToHost);
   hipMemcpy(vA, vA_d, 3*nBod*sizeof(float), hipMemcpyDeviceToHost);
   hipMemcpy(fA, fA_d, 3*nBod*sizeof(float), hipMemcpyDeviceToHost);

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);

   hipEventElapsedTime(&time, start, stop);
   printf("\nTotal time = %10.4f [sec]\n", time*1.E-3);

   free(rA);
   free(vA);
   free(fA);

   hipFree(rA_d);
   hipFree(vA_d);
   hipFree(fA_d);
   return 0;
}

// Initial conditions
void initCoord(float *rA, float *vA, float *fA, \
               float initDist, int nBod, int nI)
{
   int i, j, k;
   float Xi, Yi, Zi;
   float *rAx = &rA[     0];        //----
   float *rAy = &rA[  nBod];        // Pointers on X, Y, Z components of coordinates
   float *rAz = &rA[2*nBod];        //----
   int ii = 0;

   memset(fA, 0.0, 3*nBod*sizeof(float));
   memset(vA, 0.0, 3*nBod*sizeof(float));

   for (i = 0; i < nI; i++) {
      Xi = i*initDist;
      for (j = 0; j < nI; j++) {
         Yi = j*initDist;
         for (k = 0; k < nI; k++) {
            Zi = k*initDist;
            rAx[ii] = Xi;
            rAy[ii] = Yi;
            rAz[ii] = Zi;
            ii++;
         }
      }
   }
}

// Forces acting on each body
__global__ void forces(float *rA, float *fA, int nBod)
{
   int i, j;
   float Xi, Yi, Zi;
   float Xij, Yij, Zij;             // X[j] - X[i] and so on
   float Rij2;                      // Xij^2+Yij^2+Zij^2
   float invRij2, invRij6;          // 1/rij^2; 1/rij^6
   float *rAx = &rA[     0];        //----
   float *rAy = &rA[  nBod];        // Pointers on X, Y, Z components of coordinates
   float *rAz = &rA[2*nBod];        //----
   float *fAx = &fA[     0];        //----
   float *fAy = &fA[  nBod];        // Pointers on X, Y, Z components of forces
   float *fAz = &fA[2*nBod];        //----
   float magForce;                  // Force magnitude
   float const EPS = 1.E-10;        // Small value to prevent 0/0 if i==j

   i = blockDim.x*blockIdx.x + threadIdx.x;
   Xi = rAx[i];
   Yi = rAy[i];
   Zi = rAz[i];
   fAx[i] = 0.0;
   fAy[i] = 0.0;
   fAz[i] = 0.0;
   for (j = 0; j < nBod; j++) {
      Xij = rAx[j] - Xi;
      Yij = rAy[j] - Yi;
      Zij = rAz[j] - Zi;
      Rij2 = Xij*Xij + Yij*Yij + Zij*Zij;
      invRij2 = Rij2/((Rij2 + EPS)*(Rij2 + EPS));
      invRij6 = invRij2*invRij2*invRij2;
      magForce = 6.f*invRij2*(2.f*invRij6 - 1.f)*invRij6;
      fAx[i]+= Xij*magForce;
      fAy[i]+= Yij*magForce;
      fAz[i]+= Zij*magForce;
   }
}

// Integration of coordinates an velocities
__global__ void integration(float *rA, float *vA, float *fA, int nBod)
{
   int i;
   float const dt = 0.01;              // Time step
   float const mass = 1.0;             // mass of a body
   float const mdthalf = dt*0.5/mass;

   i = blockDim.x*blockIdx.x + threadIdx.x;

   rA[i]+= (vA[i] + fA[i]*mdthalf)*dt;

   vA[i]+= fA[i]*dt;
}
